#include "hip/hip_runtime.h"
// -----------------------------------------------------------------------------
//
// Copyright (C) The BioDynaMo Project.
// All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
//
// See the LICENSE file distributed with this work for details.
// See the NOTICE file distributed with this work for additional information
// regarding copyright ownership.
//
// -----------------------------------------------------------------------------

#include "core/gpu/displacement_op_cuda_kernel.h"
#include "core/gpu/math.h"
#include "core/shape.h"

#include "samples/common/inc/hip/hip_vector_types.h"

namespace bdm {

#define GpuErrchk(ans) \
  { GpuAssert((ans), __FILE__, __LINE__); }
inline void GpuAssert(hipError_t code, const char* file, int line,
                      bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert (error code %d): %s %s %d\n", code,
            hipGetErrorString(code), file, line);
    if (code == hipErrorInsufficientDriver) {
      printf(
          "This probably means that no CUDA-compatible GPU has been detected. "
          "Consider setting the use_opencl flag to \"true\" in the bmd.toml "
          "file to use OpenCL instead.\n");
    }
    if (abort)
      exit(code);
  }
}

inline __device__ double GetL2Distance(const double3& a, const double3& b) {
  double3 dist_array = b - a;
  return Norm(dist_array);
}

inline __device__ double Sum(const double3& a) { return a.x + a.y + a.z; }

__device__ double SquaredEuclidianDistance(const double* positions,
                                           uint32_t idx, uint32_t nidx) {
  const double dx = positions[3 * idx + 0] - positions[3 * nidx + 0];
  const double dy = positions[3 * idx + 1] - positions[3 * nidx + 1];
  const double dz = positions[3 * idx + 2] - positions[3 * nidx + 2];
  return (dx * dx + dy * dy + dz * dz);
}

__device__ int3 GetBoxCoordinates(uint32_t box_idx, uint32_t* num_boxes_axis) {
  int3 box_coord;
  box_coord.z = box_idx / (num_boxes_axis[0] * num_boxes_axis[1]);
  uint32_t remainder = box_idx % (num_boxes_axis[0] * num_boxes_axis[1]);
  box_coord.y = remainder / num_boxes_axis[0];
  box_coord.x = remainder % num_boxes_axis[0];
  return box_coord;
}

__device__ uint32_t GetBoxId(int3 bc, uint32_t* num_boxes_axis) {
  return bc.z * num_boxes_axis[0] * num_boxes_axis[1] +
         bc.y * num_boxes_axis[0] + bc.x;
}

__device__ void ForceBetweenSpheres(const double* positions,
                                    const double* diameters, uint32_t idx,
                                    uint32_t nidx, double3* result) {
  double r1 = 0.5 * diameters[idx];
  double r2 = 0.5 * diameters[nidx];
  // We take virtual bigger radii to have a distant interaction, to get a
  // desired density.
  double additional_radius = 10.0 * 0.15;
  r1 += additional_radius;
  r2 += additional_radius;

  double3 p1 = make_double3(positions, idx);
  double3 p2 = make_double3(positions, nidx);
  double3 comp = p1 - p2;
  double center_distance = Norm(comp);

  // the overlap distance (how much one penetrates in the other)
  double delta = r1 + r2 - center_distance;

  if (delta < 0) {
    return;
  }

  // to avoid a division by 0 if the centers are (almost) at the same location
  if (center_distance < 0.00000001) {
    *result += make_double3(42, 42, 42);
    return;
  }

  // the force itself
  double r = (r1 * r2) / (r1 + r2);
  double gamma = 1;  // attraction coeff
  double k = 2;      // repulsion coeff
  double f = k * delta - gamma * sqrt(r * delta);

  double module = f / center_distance;
  *result += comp * module;
}

__device__ double3 ComputeForceOfASphereOnASphere(const double3& c1, double r1,
                                                  const double3& c2,
                                                  double r2) {
  double3 comp = c1 - c2;
  double distance_between_centers = Norm(comp);
  double a = r1 + r2 - distance_between_centers;
  if (a < 0) {
    return make_double3(0.0, 0.0, 0.0);
  }
  if (distance_between_centers < 0.00000001) {
    return make_double3(42, 42, 42);
  } else {
    double module = a / distance_between_centers;
    return comp * module;
  }
}

__device__ void ForceOnACylinderFromASphere(
    const double* positions, const double* diameters,
    const double* ne_proximal_end, const double* ne_distal_end,
    const double* ne_axis, uint32_t idx, uint32_t nidx, double3* result,
    double* proportion_to_proximal_end) {
  double3 proximal_end = make_double3(ne_proximal_end, idx);
  double3 distal_end = make_double3(ne_distal_end, idx);
  double3 axis = make_double3(ne_axis, idx);

  double actual_length = Norm(axis);
  double d = diameters[idx];
  double3 c = make_double3(positions, nidx);
  double r = 0.5 * diameters[nidx];

  if (actual_length < r) {
    double rc = 0.5 * d;
    double3 dvec = (axis / actual_length) * rc;
    double3 npd = distal_end - dvec;
    *result = ComputeForceOfASphereOnASphere(npd, rc, c, r);
    return;
  }

  double3 proximal_end_closest = c - proximal_end;

  double proximal_end_closest_axis = Sum(proximal_end_closest * axis);
  double k = proximal_end_closest_axis / (actual_length * actual_length);
  double3 cc = proximal_end + (axis * k);

  if (k <= 1.0 && k >= 0.0) {
    *proportion_to_proximal_end = 1.0 - k;
  } else if (k < 0) {
    *proportion_to_proximal_end = 1.0;
    cc = proximal_end;
  } else {
    *proportion_to_proximal_end = 0.0;
    cc = distal_end;
  }

  double penetration = d / 2 + r - GetL2Distance(c, cc);
  if (penetration <= 0) {
    *result = make_double3(0.0, 0.0, 0.0);
    return;
  }
  *result = ComputeForceOfASphereOnASphere(cc, d * 0.5, c, r);
  return;
}

__device__ void ForceOnASphereFromACylinder(
    const double* positions, const double* diameters,
    const double* ne_proximal_end, const double* ne_distal_end,
    const double* ne_axis, uint32_t idx, uint32_t nidx, double3* result,
    double* proportion_to_proximal_end) {
  ForceOnACylinderFromASphere(positions, diameters, ne_proximal_end,
                              ne_distal_end, ne_axis, nidx, idx, result,
                              proportion_to_proximal_end);
}

__device__ void ForceBetweenCylinders(const double* positions,
                                      const double* diameters,
                                      const double* ne_proximal_end,
                                      uint32_t idx, uint32_t nidx,
                                      double3* result,
                                      double* proportion_to_proximal_end) {
  double3 a = make_double3(ne_proximal_end, idx);
  double3 c = make_double3(ne_proximal_end, nidx);
  double3 b = make_double3(positions, idx);
  double3 d = make_double3(positions, nidx);

  double d1 = diameters[idx];
  double d2 = diameters[nidx];

  double k = 0.5;  // part devoted to the distal node

  double p13x = a.x - c.x;
  double p13y = a.y - c.y;
  double p13z = a.z - c.z;
  double p43x = d.x - c.x;
  double p43y = d.y - c.y;
  double p43z = d.z - c.z;
  double p21x = b.x - a.x;
  double p21y = b.y - a.y;
  double p21z = b.z - a.z;

  double d1343 = p13x * p43x + p13y * p43y + p13z * p43z;
  double d4321 = p21x * p43x + p21y * p43y + p21z * p43z;
  double d1321 = p21x * p13x + p21y * p13y + p21z * p13z;
  double d4343 = p43x * p43x + p43y * p43y + p43z * p43z;
  double d2121 = p21x * p21x + p21y * p21y + p21z * p21z;

  double3 p1, p2;
  double denom = d2121 * d4343 - d4321 * d4321;

  // if the two segments are not ABSOLUTLY parallel
  if (denom > 0.000000000001) {
    double numer = d1343 * d4321 - d1321 * d4343;

    double mua = numer / denom;
    double mub = (d1343 + mua * d4321) / d4343;

    if (mua < 0) {
      p1 = a;
      k = 1;
    } else if (mua > 1) {
      p1 = b;
      k = 0;
    } else {
      p1 = make_double3(a.x + mua * p21x, a.y + mua * p21y, a.z + mua * p21z);
      k = 1 - mua;
    }

    if (mub < 0) {
      p2 = c;
    } else if (mub > 1) {
      p2 = d;
    } else {
      p2 = make_double3(c.x + mub * p43x, c.y + mub * p43y, c.z + mub * p43z);
    }

  } else {
    p1 = a + (b - a) * 0.5;
    p2 = c + (d - c) * 0.5;
  }

  // W put a virtual sphere on the two cylinders
  *result = ComputeForceOfASphereOnASphere(p1, d1 / 2.0, p2, d2 / 2.0) * 10;
  *proportion_to_proximal_end = k;
}

__device__ void GetForce(uint32_t idx, uint32_t nidx, const uint8_t* shape,
                         const double* positions, const double* diameters,
                         const double* ne_proximal_end,
                         const double* ne_distal_end, const double* ne_axis,
                         const uint32_t* daughter_left,
                         const uint32_t* daughter_right, const uint32_t* mother,
                         double3* result, double* ppe = nullptr) {
  if (shape[idx] == Shape::kSphere && shape[nidx] == Shape::kSphere) {
    ForceBetweenSpheres(positions, diameters, idx, nidx, result);
  } else if (shape[idx] == Shape::kSphere && shape[nidx] == Shape::kCylinder) {
    ForceOnASphereFromACylinder(positions, diameters, ne_proximal_end,
                                ne_distal_end, ne_axis, idx, nidx, result, ppe);
  } else if (shape[idx] == Shape::kCylinder && shape[nidx] == Shape::kSphere) {
    ForceOnACylinderFromASphere(positions, diameters, ne_proximal_end,
                                ne_distal_end, ne_axis, idx, nidx, result, ppe);
  } else if (shape[idx] == Shape::kCylinder &&
             shape[nidx] == Shape::kCylinder) {
    ForceBetweenCylinders(positions, diameters, ne_proximal_end, idx, nidx,
                          result, ppe);
  }
}

__device__ void DisplacementCylinder(
    uint32_t idx, const uint8_t* shape, const double* positions,
    const double* diameters, const double* ne_proximal_end,
    const double* ne_distal_end, const double* ne_axis,
    const uint32_t* daughter_left, const uint32_t* daughter_right,
    const uint32_t* mother, uint32_t start, uint16_t length,
    const uint32_t* successors, const double* squared_radius, double3* result,
    double3* result2, bool* has_neurite_neighbor) {
  uint32_t nidx = start;
  for (uint16_t nb = 0; nb < length; nb++) {
    if (nidx != idx) {
      if (SquaredEuclidianDistance(positions, idx, nidx) < *squared_radius) {
        // TODO: we should probably also check if there is a daughter or
        // mother with `has_daughter_or_mother`
        if (shape[nidx] == Shape::kCylinder) {
          if (daughter_left[idx] == nidx || daughter_right[idx] == nidx ||
              mother[idx] == nidx || mother[idx] == mother[nidx]) {
            return;
          }
        }
        double3 force_from_neighbor = make_double3(0, 0, 0);
        double proportion_to_proximal_end = 0;
        GetForce(idx, nidx, shape, positions, diameters, ne_proximal_end,
                 ne_distal_end, ne_axis, daughter_left, daughter_right, mother,
                 &force_from_neighbor, &proportion_to_proximal_end);

        double h_over_m = 0.01;
        if (shape[nidx] == Shape::kCylinder) {
          force_from_neighbor = force_from_neighbor * h_over_m;
          *has_neurite_neighbor = true;
        }

        if (proportion_to_proximal_end < 1E-10) {
          *result += force_from_neighbor;
        } else {
          double part_for_point_mass = 1.0 - proportion_to_proximal_end;
          *result += force_from_neighbor * part_for_point_mass;
          *result2 += force_from_neighbor * proportion_to_proximal_end;
        }
      }
    }
    // traverse linked-list
    nidx = successors[nidx];
  }
}

__device__ void DisplacementSphere(
    uint32_t idx, const uint8_t* shape, const double* positions,
    const double* diameters, const double* ne_proximal_end,
    const double* ne_distal_end, const double* ne_axis,
    const uint32_t* daughter_left, const uint32_t* daughter_right,
    const uint32_t* mother, uint32_t start, uint16_t length,
    const uint32_t* successors, const double* squared_radius, double3* result) {
  uint32_t nidx = start;
  for (uint16_t nb = 0; nb < length; nb++) {
    if (nidx != idx) {
      if (SquaredEuclidianDistance(positions, idx, nidx) < squared_radius[0]) {
        GetForce(idx, nidx, shape, positions, diameters, ne_proximal_end,
                 ne_distal_end, ne_axis, daughter_left, daughter_right, mother,
                 result);
      }
    }
  }
  // traverse linked-list
  nidx = successors[nidx];
}

/// Entry point to resolving collisions between all objects on GPU
__global__ void ResolveCollisions(
    const uint8_t* shape, const double* positions, const double* diameters,
    const double* tractor_force, const double* adherence,
    const uint32_t* box_id, const double* mass, const double* ne_proximal_end,
    const double* ne_distal_end, const double* ne_axis,
    const double* ne_tension, const double* force_to_transmit_to_proximal_mass,
    const uint32_t* daughter_left, const uint32_t* daughter_right,
    const uint32_t* mother, const uint8_t* has_daughter_or_mother,
    const double* timestep, const double* max_displacement,
    const double* squared_radius, const uint32_t* num_objects, uint32_t* starts,
    uint16_t* lengths, uint64_t* timestamps, uint64_t* current_timestamp,
    uint32_t* successors, uint32_t* box_length, uint32_t* num_boxes_axis,
    int32_t* grid_dimensions, double* result, double* result2) {
  uint32_t tidx = blockIdx.x * blockDim.x + threadIdx.x;
  if (tidx < *num_objects) {
    double3 movement_at_next_step = make_double3(0, 0, 0);
    if (shape[tidx] == Shape::kSphere) {  // If shape is spherical
      double3 translation_force_on_point_mass = make_double3(0, 0, 0);
      double3 tf =
          make_double3(tractor_force[3 * tidx + 0], tractor_force[3 * tidx + 1],
                       tractor_force[3 * tidx + 2]);

      movement_at_next_step = tf * (*timestep);

      // Moore neighborhood
      int3 box_coords = GetBoxCoordinates(box_id[tidx], num_boxes_axis);
      for (int z = -1; z <= 1; z++) {
        for (int y = -1; y <= 1; y++) {
          for (int x = -1; x <= 1; x++) {
            uint32_t bidx =
                GetBoxId(box_coords + make_int3(x, y, z), num_boxes_axis);
            if (timestamps[bidx] == *current_timestamp && lengths[bidx] != 0) {
              DisplacementSphere(
                  tidx, shape, positions, diameters, ne_proximal_end,
                  ne_distal_end, ne_axis, daughter_left, daughter_right, mother,
                  starts[bidx], lengths[bidx], successors, squared_radius,
                  &translation_force_on_point_mass);
            }
          }
        }
      }

      // Mass needs to non-zero!
      double mh = *timestep / mass[tidx];

      if (Norm(translation_force_on_point_mass) > adherence[tidx]) {
        movement_at_next_step += translation_force_on_point_mass * mh;

        if (Norm(translation_force_on_point_mass) * mh > *max_displacement) {
          movement_at_next_step = Normalize(movement_at_next_step);
          movement_at_next_step *= *max_displacement;
        }
      }
    } else {  // If shape is cylindrical
      double3 force_on_my_point_mass = make_double3(0, 0, 0);
      double3 force_on_my_mothers_point_mass = make_double3(0, 0, 0);
      double3 force_from_neighbors = make_double3(0, 0, 0);

      double3 axis = make_double3(ne_axis, tidx);
      double factor = -ne_tension[tidx] / Norm(axis);

      force_on_my_point_mass += axis * factor;

      if (has_daughter_or_mother[tidx] | kHasDaughterLeft == kHasDaughterLeft) {
        auto didx = daughter_left[tidx];
        double3 d_axis = make_double3(ne_axis, didx);
        auto f = ne_tension[didx] / Norm(d_axis);
        if (f < 0) {
          f = 0;
        }
        auto force_from_daughter =
            d_axis * f + make_double3(force_to_transmit_to_proximal_mass, didx);
        force_on_my_point_mass += force_from_daughter;
      }
      if (has_daughter_or_mother[tidx] |
          kHasDaughterRight == kHasDaughterRight) {
        auto didx = daughter_right[tidx];
        double3 d_axis = make_double3(ne_axis, didx);
        auto f = ne_tension[didx] / Norm(d_axis);
        if (f < 0) {
          f = 0;
        }
        auto force_from_daughter =
            d_axis * f + make_double3(force_to_transmit_to_proximal_mass, didx);
        force_on_my_point_mass += force_from_daughter;
      }

      double h_over_m = 0.01;
      bool has_neurite_neighbor = false;

      // Moore neighborhood
      int3 box_coords = GetBoxCoordinates(box_id[tidx], num_boxes_axis);
      for (int z = -1; z <= 1; z++) {
        for (int y = -1; y <= 1; y++) {
          for (int x = -1; x <= 1; x++) {
            uint32_t bidx =
                GetBoxId(box_coords + make_int3(x, y, z), num_boxes_axis);
            if (timestamps[bidx] == *current_timestamp && lengths[bidx] != 0) {
              DisplacementCylinder(
                  tidx, shape, positions, diameters, ne_proximal_end,
                  ne_distal_end, ne_axis, daughter_left, daughter_right, mother,
                  starts[bidx], lengths[bidx], successors, squared_radius,
                  &force_from_neighbors, &force_on_my_mothers_point_mass,
                  &has_neurite_neighbor);
            }
          }
        }
      }

      if (has_neurite_neighbor) {
        force_on_my_point_mass *= h_over_m;
      }

      force_on_my_point_mass += force_from_neighbors;
      result2[3 * tidx + 0] = force_on_my_mothers_point_mass.x;
      result2[3 * tidx + 1] = force_on_my_mothers_point_mass.y;
      result2[3 * tidx + 2] = force_on_my_mothers_point_mass.z;
      double force_norm = Norm(force_on_my_point_mass);
      if (force_norm > adherence[tidx]) {
        movement_at_next_step = force_on_my_point_mass;
        if (force_norm > *max_displacement) {
          movement_at_next_step *= *max_displacement / force_norm;
        }
      }
    }
    result[3 * tidx + 0] = movement_at_next_step.x;
    result[3 * tidx + 1] = movement_at_next_step.y;
    result[3 * tidx + 2] = movement_at_next_step.z;
  }
}

bdm::DisplacementOpCudaKernel::DisplacementOpCudaKernel(uint32_t num_objects,
                                                        uint32_t num_boxes) {
  GpuErrchk(hipMalloc(&d_positions_, 3 * num_objects * sizeof(double)));
  GpuErrchk(hipMalloc(&d_shape_, num_objects * sizeof(double)));
  GpuErrchk(hipMalloc(&d_diameters_, num_objects * sizeof(double)));
  GpuErrchk(hipMalloc(&d_tractor_force_, 3 * num_objects * sizeof(double)));
  GpuErrchk(hipMalloc(&d_adherence_, num_objects * sizeof(double)));
  GpuErrchk(hipMalloc(&d_ne_proximal_end_, 3 * num_objects * sizeof(double)));
  GpuErrchk(hipMalloc(&d_ne_distal_end_, 3 * num_objects * sizeof(double)));
  GpuErrchk(hipMalloc(&d_ne_axis_, 3 * num_objects * sizeof(double)));
  GpuErrchk(hipMalloc(&d_ne_tension_, num_objects * sizeof(double)));
  GpuErrchk(hipMalloc(&d_ne_force_to_transmit_to_proximal_mass_,
                       3 * num_objects * sizeof(double)));
  GpuErrchk(hipMalloc(&d_daughter_left_, num_objects * sizeof(uint32_t)));
  GpuErrchk(hipMalloc(&d_daughter_right_, num_objects * sizeof(uint32_t)));
  GpuErrchk(hipMalloc(&d_mother_, num_objects * sizeof(uint32_t)));
  GpuErrchk(
      hipMalloc(&d_has_daughter_or_mother_, num_objects * sizeof(uint8_t)));
  GpuErrchk(hipMalloc(&d_box_id_, num_objects * sizeof(uint32_t)));
  GpuErrchk(hipMalloc(&d_mass_, num_objects * sizeof(double)));
  GpuErrchk(hipMalloc(&d_timestep_, sizeof(double)));
  GpuErrchk(hipMalloc(&d_max_displacement_, sizeof(double)));
  GpuErrchk(hipMalloc(&d_squared_radius_, sizeof(double)));
  GpuErrchk(hipMalloc(&d_num_objects_, sizeof(uint32_t)));
  GpuErrchk(hipMalloc(&d_starts_, num_boxes * sizeof(uint32_t)));
  GpuErrchk(hipMalloc(&d_lengths_, num_boxes * sizeof(uint16_t)));
  GpuErrchk(hipMalloc(&d_timestamps_, num_boxes * sizeof(uint64_t)));
  GpuErrchk(hipMalloc(&d_current_timestamp_, sizeof(uint64_t)));
  GpuErrchk(hipMalloc(&d_successors_, num_objects * sizeof(uint32_t)));
  GpuErrchk(hipMalloc(&d_box_length_, sizeof(uint32_t)));
  GpuErrchk(hipMalloc(&d_num_boxes_axis_, 3 * sizeof(uint32_t)));
  GpuErrchk(hipMalloc(&d_grid_dimensions_, 3 * sizeof(int32_t)));
  GpuErrchk(hipMalloc(&d_cell_movements_, 3 * num_objects * sizeof(double)));
  GpuErrchk(hipMalloc(&d_force_to_transmit_to_proximal_mass_,
                       3 * num_objects * sizeof(double)));
}

void bdm::DisplacementOpCudaKernel::LaunchDisplacementKernel(
    const uint8_t* shape, const double* positions, const double* diameters,
    const double* tractor_force, const double* adherence,
    const uint32_t* box_id, const double* mass, const double* ne_proximal_end,
    const double* ne_distal_end, const double* ne_axis,
    const double* ne_tension,
    const double* ne_force_to_transmit_to_proximal_mass,
    const uint32_t* daughter_left, const uint32_t* daughter_right,
    const uint32_t* mother, const uint8_t* has_daughter_or_mother,
    const double* timestep, const double* max_displacement,
    const double* squared_radius, const uint32_t* num_objects, uint32_t* starts,
    uint16_t* lengths, uint64_t* timestamps, uint64_t* current_timestamp,
    uint32_t* successors, uint32_t* box_length, uint32_t* num_boxes_axis,
    int32_t* grid_dimensions, double* cell_movements,
    double* force_to_transmit_to_proximal_mass) {
  uint32_t num_boxes =
      num_boxes_axis[0] * num_boxes_axis[1] * num_boxes_axis[2];

  GpuErrchk(hipMemcpy(d_shape_, shape,
        num_objects[0] * sizeof(uint8_t),
        hipMemcpyHostToDevice));
  GpuErrchk(hipMemcpy(d_positions_, positions,
                       3 * num_objects[0] * sizeof(double),
                       hipMemcpyHostToDevice));
  GpuErrchk(hipMemcpy(d_diameters_, diameters, num_objects[0] * sizeof(double),
                       hipMemcpyHostToDevice));
  GpuErrchk(hipMemcpy(d_tractor_force_, tractor_force,
                       3 * num_objects[0] * sizeof(double),
                       hipMemcpyHostToDevice));
  GpuErrchk(hipMemcpy(d_adherence_, adherence, num_objects[0] * sizeof(double),
                       hipMemcpyHostToDevice));
  GpuErrchk(hipMemcpy(d_ne_proximal_end_, ne_proximal_end,
                       3 * num_objects[0] * sizeof(double),
                       hipMemcpyHostToDevice));
  GpuErrchk(hipMemcpy(d_ne_distal_end_, ne_distal_end,
                       3 * num_objects[0] * sizeof(double),
                       hipMemcpyHostToDevice));
  GpuErrchk(hipMemcpy(d_ne_axis_, ne_axis, 3 * num_objects[0] * sizeof(double),
                       hipMemcpyHostToDevice));
  GpuErrchk(hipMemcpy(d_ne_tension_, ne_tension,
                       num_objects[0] * sizeof(double),
                       hipMemcpyHostToDevice));
  GpuErrchk(hipMemcpy(d_ne_force_to_transmit_to_proximal_mass_,
                       ne_force_to_transmit_to_proximal_mass,
                       3 * num_objects[0] * sizeof(double),
                       hipMemcpyHostToDevice));
  GpuErrchk(hipMemcpy(d_daughter_left_, daughter_left,
                       num_objects[0] * sizeof(uint32_t),
                       hipMemcpyHostToDevice));
  GpuErrchk(hipMemcpy(d_daughter_right_, daughter_right,
                       num_objects[0] * sizeof(uint32_t),
                       hipMemcpyHostToDevice));
  GpuErrchk(hipMemcpy(d_mother_, mother, num_objects[0] * sizeof(uint32_t),
                       hipMemcpyHostToDevice));
  GpuErrchk(hipMemcpy(d_has_daughter_or_mother_, has_daughter_or_mother,
                       num_objects[0] * sizeof(uint8_t),
                       hipMemcpyHostToDevice));
  GpuErrchk(hipMemcpy(d_box_id_, box_id, num_objects[0] * sizeof(uint32_t),
                       hipMemcpyHostToDevice));
  GpuErrchk(hipMemcpy(d_mass_, mass, num_objects[0] * sizeof(double),
                       hipMemcpyHostToDevice));
  GpuErrchk(hipMemcpy(d_timestep_, timestep, sizeof(double),
                       hipMemcpyHostToDevice));
  GpuErrchk(hipMemcpy(d_max_displacement_, max_displacement, sizeof(double),
                       hipMemcpyHostToDevice));
  GpuErrchk(hipMemcpy(d_squared_radius_, squared_radius, sizeof(double),
                       hipMemcpyHostToDevice));
  GpuErrchk(hipMemcpy(d_num_objects_, num_objects, sizeof(uint32_t),
                       hipMemcpyHostToDevice));
  GpuErrchk(hipMemcpy(d_starts_, starts, num_boxes * sizeof(uint32_t),
                       hipMemcpyHostToDevice));
  GpuErrchk(hipMemcpy(d_lengths_, lengths, num_boxes * sizeof(uint16_t),
                       hipMemcpyHostToDevice));
  GpuErrchk(hipMemcpy(d_timestamps_, timestamps, num_boxes * sizeof(uint64_t),
                       hipMemcpyHostToDevice));
  GpuErrchk(hipMemcpy(d_current_timestamp_, current_timestamp,
                       sizeof(uint64_t), hipMemcpyHostToDevice));
  GpuErrchk(hipMemcpy(d_successors_, successors,
                       num_objects[0] * sizeof(uint32_t),
                       hipMemcpyHostToDevice));
  GpuErrchk(hipMemcpy(d_box_length_, box_length, sizeof(uint32_t),
                       hipMemcpyHostToDevice));
  GpuErrchk(hipMemcpy(d_num_boxes_axis_, num_boxes_axis, 3 * sizeof(uint32_t),
                       hipMemcpyHostToDevice));
  GpuErrchk(hipMemcpy(d_grid_dimensions_, grid_dimensions,
                       3 * sizeof(uint32_t), hipMemcpyHostToDevice));

  int blockSize = 128;
  int minGridSize;
  int gridSize;

  // Get a near-optimal occupancy with the following thread organization
  hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize,
                                     ResolveCollisions, 0, num_objects[0]);
  gridSize = (num_objects[0] + blockSize - 1) / blockSize;

  // printf("gridSize = %d  |  blockSize = %d\n", gridSize, blockSize);
  ResolveCollisions<<<gridSize, blockSize>>>(
      d_shape_, d_positions_, d_diameters_, d_tractor_force_, d_adherence_,
      d_box_id_, d_mass_, d_ne_proximal_end_, d_ne_distal_end_, d_ne_axis_,
      d_ne_tension_, d_ne_force_to_transmit_to_proximal_mass_, d_daughter_left_,
      d_daughter_right_, d_mother_, d_has_daughter_or_mother_, d_timestep_,
      d_max_displacement_, d_squared_radius_, d_num_objects_, d_starts_,
      d_lengths_, d_timestamps_, d_current_timestamp_, d_successors_,
      d_box_length_, d_num_boxes_axis_, d_grid_dimensions_, d_cell_movements_,
      d_force_to_transmit_to_proximal_mass_);

  // We need to wait for the kernel to finish before reading back the result
  hipDeviceSynchronize();
  hipMemcpy(cell_movements, d_cell_movements_,
             3 * num_objects[0] * sizeof(double), hipMemcpyDeviceToHost);
}

void bdm::DisplacementOpCudaKernel::ResizeCellBuffers(uint32_t num_cells) {
  hipFree(d_shape_);
  hipFree(d_positions_);
  hipFree(d_diameters_);
  hipFree(d_tractor_force_);
  hipFree(d_adherence_);
  hipFree(d_ne_proximal_end_);
  hipFree(d_ne_distal_end_);
  hipFree(d_ne_axis_);
  hipFree(d_ne_tension_);
  hipFree(d_ne_force_to_transmit_to_proximal_mass_);
  hipFree(d_daughter_left_);
  hipFree(d_daughter_right_);
  hipFree(d_mother_);
  hipFree(d_has_daughter_or_mother_);
  hipFree(d_box_id_);
  hipFree(d_mass_);
  hipFree(d_successors_);
  hipFree(d_cell_movements_);
  hipFree(d_force_to_transmit_to_proximal_mass_);

  hipMalloc(&d_shape_, num_cells * sizeof(uint8_t));
  hipMalloc(&d_positions_, 3 * num_cells * sizeof(double));
  hipMalloc(&d_diameters_, num_cells * sizeof(double));
  hipMalloc(&d_tractor_force_, 3 * num_cells * sizeof(double));
  hipMalloc(&d_ne_proximal_end_, 3 * num_cells * sizeof(double));
  hipMalloc(&d_ne_distal_end_, 3 * num_cells * sizeof(double));
  hipMalloc(&d_ne_axis_, 3 * num_cells * sizeof(double));
  hipMalloc(&d_ne_tension_, num_cells * sizeof(double));
  hipMalloc(&d_ne_force_to_transmit_to_proximal_mass_,
             3 * num_cells * sizeof(double));
  hipMalloc(&d_adherence_, num_cells * sizeof(double));
  hipMalloc(&d_daughter_left_, num_cells * sizeof(uint32_t));
  hipMalloc(&d_daughter_right_, num_cells * sizeof(uint32_t));
  hipMalloc(&d_mother_, num_cells * sizeof(uint32_t));
  hipMalloc(&d_has_daughter_or_mother_, num_cells * sizeof(uint32_t));
  hipMalloc(&d_box_id_, num_cells * sizeof(uint32_t));
  hipMalloc(&d_mass_, num_cells * sizeof(double));
  hipMalloc(&d_successors_, num_cells * sizeof(uint32_t));
  hipMalloc(&d_cell_movements_, 3 * num_cells * sizeof(double));
  hipMalloc(&d_force_to_transmit_to_proximal_mass_,
             3 * num_cells * sizeof(double));
}

void bdm::DisplacementOpCudaKernel::ResizeGridBuffers(uint32_t num_boxes) {
  hipFree(d_starts_);
  hipFree(d_lengths_);
  hipFree(d_timestamps_);

  hipMalloc(&d_starts_, num_boxes * sizeof(uint32_t));
  hipMalloc(&d_lengths_, num_boxes * sizeof(uint16_t));
  hipMalloc(&d_timestamps_, num_boxes * sizeof(uint64_t));
}

bdm::DisplacementOpCudaKernel::~DisplacementOpCudaKernel() {
  hipFree(d_shape_);
  hipFree(d_positions_);
  hipFree(d_diameters_);
  hipFree(d_tractor_force_);
  hipFree(d_adherence_);
  hipFree(d_box_id_);
  hipFree(d_mass_);
  hipFree(d_timestep_);
  hipFree(d_max_displacement_);
  hipFree(d_squared_radius_);
  hipFree(d_num_objects_);
  hipFree(d_starts_);
  hipFree(d_lengths_);
  hipFree(d_timestamps_);
  hipFree(d_current_timestamp_);
  hipFree(d_successors_);
  hipFree(d_num_boxes_axis_);
  hipFree(d_grid_dimensions_);
  hipFree(d_cell_movements_);
  hipFree(d_ne_proximal_end_);
  hipFree(d_ne_distal_end_);
  hipFree(d_ne_axis_);
  hipFree(d_ne_tension_);
  hipFree(d_ne_force_to_transmit_to_proximal_mass_);
  hipFree(d_force_to_transmit_to_proximal_mass_);
  hipFree(d_daughter_left_);
  hipFree(d_daughter_right_);
  hipFree(d_mother_);
  hipFree(d_has_daughter_or_mother_);
}

}  // namespace bdm
